/*
 * Nvidia Tesla C2075 GPU
 * Jens Lang, 2013
 * This programme creates high-resolution power profiles for GPU routines executed on Nvidia GPUs.
 * It needs the CUDA, NVML and PAPI libraries. It should be compiled with gcc using the switch
 * -std=c++11. For further information, please refer to:
 * Lang, Jens; Rünger, Gudula: High-Resolution Power Profiling of GPU Functions Using Low-Resolution
 * Measurement. In: Wolf, F.; Mohr, B.; an Mey, D. (Hrsg.): Euro-Par 2013 Parallel Processing
 * (LNCS, Bd. 8097): S. 801–812. Springer  –  ISBN 978-3-642-40046-9, 2013. DOI: 10.1007/978-3-642-40047-6_80
 */
#include <papi.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>

#include <hip/hip_runtime.h>
#include "hipblas.h"
#include "nvml.h"

#include "gpu_heater.h"

//#define M 2300
int M = 19456;
//int M = 1024;
#define IDX2C(i,j,M) (i*M+j)

hipblasHandle_t handle;

typedef float matrix_t;
matrix_t* devPtrA;
matrix_t* devPtrB;
matrix_t* devPtrC;

void heatup_gpu() 
{
	// cuBLAS call
	matrix_t alpha = 2.3f;
	matrix_t beta = 5.7f;

	for (int xxx=0; xxx<1; xxx++) {
		hipblasSgemm(
			handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
			M, M, M,
			&alpha,
			devPtrA, M,
			devPtrB, M,
			&beta,
			devPtrC, M);

		hipDeviceSynchronize();	// Wait for the GPU launched work to complete
	}
//	status = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha,
//	d_A, N, d_B, N, &beta, d_C, N);
}

void init_heater()
{

	/* Initialise random number generator (for sleep) */
/*	std::uniform_int_distribution<long long> random_sleep_time{20*1000*1000, 40*1000*1000};
	std::default_random_engine random_engine(std::chrono::system_clock::now().time_since_epoch().count());
*/
	hipError_t cudaStat;
	hipblasStatus_t stat;

	// allocate host memory for matrices and copy them to the device
	int i, j;
	matrix_t* a;
	matrix_t* b;
	//matrix_t* c;
	a = (matrix_t *)malloc (M * M * sizeof (*a));
	b = (matrix_t *)malloc (M * M * sizeof (*b));
	//c = (matrix_t *)malloc (M * M * sizeof (*c));
	if (!a) {
			printf ("host memory allocation failed\n");
			return;
	}
	for (j = 0; j < M; j++) {
			for (i = 0; i < M; i++) {
					a[IDX2C(i,j,M)] = (matrix_t)(i * M + j + 1);
					b[IDX2C(i,j,M)] = (matrix_t)(i * M + j - 1);
			}
	}
	cudaStat = hipMalloc ((void**)&devPtrA, M*M*sizeof(*a));
	if (cudaStat != hipSuccess) {
			printf ("device memory allocation failed\n");
			return;
	}
	cudaStat = hipMalloc ((void**)&devPtrB, M*M*sizeof(*b));
	if (cudaStat != hipSuccess) {
			printf ("device memory allocation failed\n");
			return;
	}
	cudaStat = hipMalloc ((void**)&devPtrC, M*M*sizeof(*b));
	if (cudaStat != hipSuccess) {
			printf ("device memory allocation failed\n");
			return;
	}
	stat = hipblasCreate(&handle);
	if (stat != HIPBLAS_STATUS_SUCCESS) {
			printf ("CUBLAS initialization failed\n");
			return;
	}
	stat = hipblasSetMatrix (M, M, sizeof(*a), a, M, devPtrA, M);
	if (stat != HIPBLAS_STATUS_SUCCESS) {
			printf ("data download failed\n");
			hipFree (devPtrA);
			hipblasDestroy(handle);
			return;
	}
	stat = hipblasSetMatrix (M, M, sizeof(*b), b, M, devPtrB, M);
	if (stat != HIPBLAS_STATUS_SUCCESS) {
			printf ("data download failed\n");
			hipFree (devPtrB);
			hipblasDestroy(handle);
			return;
	}

  printf("#Size of matrices: %d\n", M);
}

void shutdown_heater() 
{
	// shutdown cuBLAS
	hipFree (devPtrA);
	hipblasDestroy(handle);
}
