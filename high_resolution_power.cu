/*
 * This program is an extended version of "Jens Lang"'s work.  
 * Lang, J., & Rünger, G. (2013). High-Resolution power profiling of GPU
 * functions using low-resolution measurement. In Euro-Par 2013 Parallel
 * Processing (pp. 801-812). Springer Berlin Heidelberg.
 */

#include <papi.h>
#include <nvml.h>
#include <stdio.h>
#include <unistd.h>
#include <pthread.h>

#include "high_resolution_power.h"

#define CEIL(x,y) 1 + (((x) - 1) / (y))

int sleep_time_after_kernel_call = 0; //in seconds
int reps = 1; //getenv
unsigned int device_id = 0;
long long time_kernel_start_ex = 0;

long long kernel_execution_time = 0; //nano seconds

nvmlReturn_t nvml_result;
nvmlDevice_t nvml_device;


void set_reps(int nvml_reps) {
  reps = nvml_reps;
} 

void handle_papi_error(int retval)
{
	PAPI_perror((char *) "Fehler");
	printf((char *) "PAPI error %d: %s\n", retval, PAPI_strerror(retval));
}

int nvml_finalize(nvmlReturn_t result) {
  result = nvmlShutdown();
  if (NVML_SUCCESS != result) {
    printf("Failed to shutdown NVML: %s\n", nvmlErrorString(result));
    return -1;
  }
  return 1;
}

unsigned int synchronizeTime(long long & cpu_time, nvmlDevice_t nvml_device, unsigned int &temperature, int &pstate)
{
	unsigned int gpu_value = 0;
	unsigned int last_gpu_value = 0;


  int device_id = 0;
  long long starttt = PAPI_get_real_nsec();
	nvmlReturn_t nvml_result;
	nvml_result = nvmlDeviceGetPowerUsage(nvml_device, &gpu_value);
	if (nvml_result != NVML_SUCCESS) printf("NVML error: %s.\n", nvmlErrorString(nvml_result));
	last_gpu_value = gpu_value;

	while (gpu_value == last_gpu_value)
	{
		last_gpu_value = gpu_value;
		nvml_result = nvmlDeviceGetPowerUsage(nvml_device, &gpu_value);
		if (nvml_result != NVML_SUCCESS) printf("NVML error: %d.\n", nvmlErrorString(nvml_result));
	}

	cpu_time = PAPI_get_real_nsec();
  
///////////////////////////////////////
  nvml_result = nvmlDeviceGetTemperature(nvml_device, NVML_TEMPERATURE_GPU, &temperature);
  if (NVML_SUCCESS != nvml_result)
  {
    printf("Failed to get temperature of the device %i: %s\n", device_id, nvmlErrorString(nvml_result));
  }
  nvmlPstates_t p_state;
  nvml_result = nvmlDeviceGetPerformanceState(nvml_device, &p_state);
  if (NVML_SUCCESS != nvml_result)
  {
    printf("Failed to get perf state of the device %i: %s\n", device_id, nvmlErrorString(nvml_result));
  }

  pstate = p_state;
//////////////////////////////////////

	return gpu_value;
}


int nvml_init() { 
  char *nvml_reps = getenv("NVML_REPS");
  if (nvml_reps != NULL) {
    reps = atoi(nvml_reps);
  }

  char *nvml_sleep_time = getenv("NVML_SLEEP_TIME");
  if (nvml_sleep_time != NULL) {
    sleep_time_after_kernel_call = atoi(nvml_sleep_time);
  }

	// Initialize the PAPI library
	int retval = PAPI_NULL;
  retval = PAPI_library_init(PAPI_VER_CURRENT);
	if (retval != PAPI_VER_CURRENT) {
		printf("PAPI library init error: %d!\n", retval);
    handle_papi_error(retval);
		return -1;
	} 

  //initializing nvml
  nvml_result = nvmlInit();
  if (NVML_SUCCESS != nvml_result)
  {
    printf("Failed to initialize NVML: %s\n", nvmlErrorString(nvml_result));
    return -1;
  }

	nvml_result = nvmlDeviceGetHandleByIndex(device_id, &nvml_device);
  if (NVML_SUCCESS != nvml_result)
  {
    printf("Failed to get handle for device %i: %s\n", device_id, nvmlErrorString(nvml_result));
    nvml_result = nvmlShutdown();
    nvml_finalize(nvml_result);
    return -1;
  }
  
  printf("#defined #NUM_REPS: %d COOLDOWN_TIME: %ds\n", reps, sleep_time_after_kernel_call);

  return 0;
}


int power_profile(void *(*call_cuda_kernel)(void*), void *(*reset_kenel_data)(void*)) {
  struct timespec sleep_time;
  sleep_time.tv_sec = sleep_time_after_kernel_call;
  sleep_time.tv_nsec = 0;

  pthread_t pthread;

	reps = CEIL(7E9, kernel_execution_time);
	
	printf("#RREPS: %d\n", reps);

  // perform actual energy measurement
	for (int n_ex = 0; n_ex < reps; n_ex++)
	{
    reset_kenel_data(NULL);
		// wait a random time // this functionality is suspended since we just
		// interested in power after 5seconds
	/*	sleep_time.tv_nsec = rand()%(DELTA_T) + DELTA_T;
		nanosleep(&sleep_time, NULL);
*/
		// call GPU kernel
		bool gpu_power_before_printed = false; // indicates wether the first power measurement (before the start of the GPU kernel) has already been printed out
		long long time_start_kernel = PAPI_get_real_nsec();
		long long time_current = time_start_kernel;
		if (n_ex == 0) {
			time_kernel_start_ex = time_start_kernel;
		}

    int rc = pthread_create(&pthread, NULL, call_cuda_kernel, NULL);
    if (rc){
     printf("ERROR; return code from pthread_create() is %d\n", rc);
     exit(-1);
    }

		unsigned int gpu_power_before;
		nvml_result = nvmlDeviceGetPowerUsage(nvml_device, &gpu_power_before);
		if (nvml_result != NVML_SUCCESS) {
      printf("NVML error: %s.\n", nvmlErrorString(nvml_result));
    }

    
		// continually retrieve power values of the GPU
		while (time_current < time_start_kernel + kernel_execution_time)
		{
			long long time_running_update;
      unsigned int temperature;
      int pstate;
			unsigned int gpu_power = synchronizeTime(time_running_update, nvml_device, temperature, pstate);

			if (!gpu_power_before_printed)
			{
				printf("%.5f ms\t%.5f W\t%u C\t%d\t%.5f ms\n", (time_running_update - time_start_kernel - DELTA_T) / 1e6, gpu_power_before / 1e3, temperature, pstate, (time_current - time_kernel_start_ex - DELTA_T) / 1e6);
				gpu_power_before_printed = true;
			}

			printf("%.5f ms\t%.5f W\t%u C\t%d\t%.5f ms\n", (time_running_update - time_start_kernel) / 1e6, gpu_power / 1e3, temperature, pstate, (time_current - time_kernel_start_ex) / 1e6);
			time_current = time_running_update;
		}

     pthread_join( pthread, NULL);
    
	}

	long long time_simulation_end;
  unsigned int t;
  int p;
	synchronizeTime(time_simulation_end, nvml_device, t, p);
	printf("# end time: %.5f\n", (time_simulation_end - time_kernel_start_ex) / 1e6);

  return 0;
}


int high_resolution_power_profile(void *(*call_cuda_kernel)(void*), void *(*reset_kenel_data)(void*)) {
  int error = nvml_init();
  if (error != 0) {
    return error;
  }
//---------------------------------------------------------------------------------------------------------------------------------
  //set kernel execution time
  reset_kenel_data(NULL);
  time_kernel_start_ex = PAPI_get_real_nsec();
  call_cuda_kernel(NULL);
  hipDeviceSynchronize();
  long long time_kernel_finish_ex = PAPI_get_real_nsec();
  kernel_execution_time = time_kernel_finish_ex - time_kernel_start_ex; 

  printf("#Kernel execution time: %.5fms\n", kernel_execution_time/1e6);
//---------------------------------------------------------------------------------------------------------------------------------
  reset_kenel_data(NULL);
  return power_profile(call_cuda_kernel, reset_kenel_data);  
}

// in nanoseconds
long long get_exec_time_in_nanoseconds(void *(*call_cuda_kernel)(void*), void *(*reset_kenel_data)(void*)) {
   //get kernel execution time
  reset_kenel_data(NULL);
  time_kernel_start_ex = PAPI_get_real_nsec();
  call_cuda_kernel(NULL);
  hipDeviceSynchronize();
  long long time_kernel_finish_ex = PAPI_get_real_nsec();
  kernel_execution_time = time_kernel_finish_ex - time_kernel_start_ex;

  return kernel_execution_time;

}
