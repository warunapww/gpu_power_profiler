#include <assert.h>
#include <stdio.h>
#include "jacobi_kernel.hu"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#include <assert.h>
#define TIME
#define N 4096
#define T 8192
float A [2][N][N];

#include <unistd.h>
#include <sys/time.h>

#include "high_resolution_power.h"

#ifdef TIME
#define IF_TIME(foo) foo;
#else
#define IF_TIME(foo)
#endif

float *dev_A;

void *call_gpu_functions(void *nothing);
void *gpu_data_reset(void *nothing);


void init_array()
{
    int i, j, k;

    for (i=0; i<N; i++) {
    for (j=0; j<N; j++)
	for (k = 0; k < 2; k++) {
            A [k][i][j] = ((float) i*(j+2) + 2) / N;
	}
    }
}


void print_array()
{
    int i, j;
    for (i=0; i<N; i++) {
    	for (j=0; j<N; j++)
        	fprintf(stdout, "%0.15lf ", A [0][i][j]);
        fprintf(stdout, "\n");
    }
    fprintf(stdout, "\n");
}

double rtclock()
{
    struct timezone Tzp;
    struct timeval Tp;
    int stat;
    stat = gettimeofday (&Tp, &Tzp);
    if (stat != 0) printf("Error return from gettimeofday: %d",stat);
    return(Tp.tv_sec + Tp.tv_usec*1.0e-6);
}



int main(int argc, char **argv)
{
    long t, i, j;
    double t_start, t_end;

    init_array();
     #ifdef __HIPCC__
	// Initialize cuda before starting the timing.
        float *dev_X;
        hipMalloc((void **) &dev_X, 1);
     #endif

    IF_TIME(t_start = rtclock());
	
#ifdef P4A
#pragma scop
    for (t=0; t < T/2; t++) {
      for (i=1; i < N-1; i++)
	#pragma ivdep
      	for (j=1; j < N-1; j++)
A[1][i][j] = (0.2f) * (A[0][i][j] + A[0][i][j-1] + A[0][i][j+1] + A[0][i+1][j] + A[0][i-1][j] );
      for (i=1; i < N-1; i++)
	#pragma ivdep
      	for (j=1; j < N-1; j++)
A[0][i][j] = (0.2f) * (A[1][i][j] + A[1][i][j-1] + A[1][i][j+1] + A[1][i+1][j] + A[1][i-1][j] );

    }
#pragma endscop

#else
{
#define cudaCheckReturn(ret) \
  do { \
    hipError_t cudaCheckReturn_e = (ret); \
    if (cudaCheckReturn_e != hipSuccess) { \
      fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(cudaCheckReturn_e)); \
      fflush(stderr); \
    } \
    assert(cudaCheckReturn_e == hipSuccess); \
  } while(0)
#define cudaCheckKernel() \
  do { \
    cudaCheckReturn(hipGetLastError()); \
  } while(0)

  
  cudaCheckReturn(hipMalloc((void **) &dev_A, (2) * (4096) * (4096) * sizeof(float)));
  
  cudaCheckReturn(hipMemcpy(dev_A, A, (2) * (4096) * (4096) * sizeof(float), hipMemcpyHostToDevice));
  
  #define floord(n,d) (((n)<0) ? -((-(n)+(d)-1)/(d)) : (n)/(d))
	
	//if you need to measure time explicitely, uncomment next two lines
	//long long exec_time_ns = get_exec_time_in_nanoseconds(call_gpu_functions, gpu_data_reset);
	//printf("Execution time: %fs\n", exec_time_ns/1E9);

	high_resolution_power_profile(call_gpu_functions, gpu_data_reset);

  cudaCheckReturn(hipMemcpy(A, dev_A, (2) * (4096) * (4096) * sizeof(float), hipMemcpyDeviceToHost));
  
  cudaCheckReturn(hipFree(dev_A));
}
#endif
	if (argc == 42)
        print_array();

    IF_TIME(t_end = rtclock());

#ifndef NGFLOPS
    IF_TIME(fprintf(stderr, "%0.3lfs, %f GFLOPS\n", t_end - t_start, 1.0e-9*9*T*(N-2)*(N-2)/(t_end-t_start)));
#else
    IF_TIME(fprintf(stderr, "%0.3lf\n", t_end - t_start, 1.0e-9*9*T*(N-2)*(N-2)/(t_end-t_start)));
#endif

    if (fopen(".test", "r")) {
        print_array();
    }
    return 0;
}

void *call_gpu_functions(void *nothing) {
	for (int h0 = 0; h0 <= 2048; h0 += 1) {
	{
		dim3 k0_dimBlock(128, 1);
		dim3 k0_dimGrid(342);
		kernel0_1 <<<k0_dimGrid, k0_dimBlock>>> (dev_A, h0);
		cudaCheckKernel();
	}
	
	if (h0 <= 2047)
	{
		dim3 k1_dimBlock(128, 1);
		dim3 k1_dimGrid(342);
		kernel1_1 <<<k1_dimGrid, k1_dimBlock>>> (dev_A, h0);
		cudaCheckKernel();
	}
		
  }
	return NULL;	
}


void *gpu_data_reset(void *nothing) {
	// for jacobi, you don't have to do anything here
	return NULL;
}
